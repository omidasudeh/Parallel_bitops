#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "read_bmp.h"
#include "read_bmp_clib.h"
// the kernel
__global__ 
void sobel(uint8_t *bmp_data,uint8_t *new_bmp_img,int threshold, int ht, int wd) 
{
	int block_origin_x = blockDim.x*blockIdx.x;
	int block_origin_y = blockDim.y*blockIdx.y;
	int pixelX = block_origin_y+threadIdx.x;
	int pixelY = block_origin_x+threadIdx.y;
	if(pixelX>0 && pixelX<wd-1 && pixelY>0 && pixelY<ht-1) // if not border
	{

		int Gx = bmp_data[(pixelY-1)*wd +(pixelX+1)] - bmp_data[(pixelY-1)*wd+(pixelX-1)] +
		       2*bmp_data[(pixelY)  *wd +(pixelX+1)] - 2*bmp_data[(pixelY)  *wd +(pixelX-1)] +
			 bmp_data[(pixelY+1)*wd +(pixelX+1)] - bmp_data[(pixelY+1)*wd+(pixelX-1)];

		int Gy = +bmp_data[(pixelY-1)*wd+(pixelX-1)] + 2*bmp_data[(pixelY-1)*wd+(pixelX)] + bmp_data[(pixelY-1)*wd+(pixelX+1)]
		  -bmp_data[(pixelY+1)*wd+(pixelX-1)] - 2*bmp_data[(pixelY+1)*wd+(pixelX)] - bmp_data[(pixelY+1)*wd +(pixelX+1)];	 

		float mag = sqrtf(Gx*Gx+Gy*Gy);
		if(mag>threshold)
			new_bmp_img[(pixelY)*wd+(pixelX)] = 255;
		else
			new_bmp_img[(pixelY)*wd+(pixelX)] = 0;
	}

}
int main(int argc,char** argv)
{
	char* input_img_file = argv[1];
	char* serial_img = argv[2];
	char* cuda_img = argv[3];
	//============== reading the binary bmp file into buffer ==============
	FILE *input_file;
	input_file = fopen(input_img_file,"rb");
	//=====================================================================	
	printf("**********************************************************************\n");
	printf("please wait. The serial version may take up to a minute to run\n");
	//1. Host memory allocation and getting the image and its attributes	
	bmp_image img;
	uint8_t *bmp_data;
	bmp_data = (uint8_t *)img.read_bmp_file(input_file);
	// allocate new output buffer of the same size for the sobel image
	uint8_t *new_bmp_img;
	new_bmp_img = (uint8_t *) malloc(img.num_pixel);
	//get image attributes
	int wd = img.image_width;	int ht = img.image_height;
//############################################## Serial Code #################################################################
	//############# convergence loop #############################
	int threshold = 0;
	int black_cell_count = 0;
	clock_t t;
	t = clock();
	while(black_cell_count<(75*wd*ht/100))
	{
		black_cell_count = 0;
		threshold+=1;
		for(int i = 1;i<(ht-1);i++)
		{
			for(int j=1;j<(wd-1);j++)
			{
				int Gx = bmp_data[(i-1)*wd +(j+1)] - bmp_data[(i-1)*wd +(j-1)] +
					2*bmp_data[(i)*wd +(j+1)] - 2*bmp_data[(i)*wd +(j-1)] +
					bmp_data[(i+1)*wd +(j+1)] - bmp_data[(i+1)*wd +(j-1)];

				int Gy = bmp_data[(i-1)*wd +(j-1)] + 2*bmp_data[(i-1)*wd +(j)] + bmp_data[(i-1)*wd +(j+1)]
					-bmp_data[(i+1)*wd +(j-1)] - 2*bmp_data[(i+1)*wd +(j)] - bmp_data[(i+1)*wd +(j+1)];	 
				float mag = sqrt(Gx*Gx+Gy*Gy);
				if(mag>threshold)
					new_bmp_img[i*wd+j] = 255;
				else
				{
					new_bmp_img[i*wd+j] = 0;
					black_cell_count++;
				}				
			}
		}
	}
	t = clock()-t;
	printf("Time taken for serial sobel operation:%f sec\n",((float)t)/CLOCKS_PER_SEC);
	printf("Threshold during convergence:%d\n\n",threshold);

	//write back the new bmp image into serial output file
	FILE *output_file;
	output_file = fopen(serial_img,"wb");
	img.write_bmp_file(output_file, new_bmp_img);
	bmp_data = (uint8_t *)img.read_bmp_file(input_file);
//############################################## CUDA Code ###################################################################
	// allocate new output buffer of the same size for the sobel image
	uint8_t *new_bmp_img1;
	new_bmp_img1 = (uint8_t *) malloc(img.num_pixel);
	//2. define thread hirearchy
	// the Grid has 32*32 blocks
	int GridDimX = ceil((float)ht/32);// Grid X dimension 
	int GridDimY = ceil((float)wd/32);// Grid Y dimension
	// each block has 32*32 threads;the load balancing is one pixel per thread	
	int BlockDimX = 32;  
	int BlockDimY = 32;
	
	//3. Device Memory allocation 
	uint8_t *bmp_data_device;//pointer to device version of the bmp_data,
	uint8_t *new_bmp_img_device;//pointer to device version of the output sobel image new_bmp_img
	hipMalloc( (void**)&bmp_data_device, img.num_pixel);
	hipMalloc( (void**)&new_bmp_img_device, img.num_pixel);
	
	//4. copy bmp_data,black_cell_count_array_host to the device
	hipMemcpy( bmp_data_device, bmp_data, img.num_pixel, hipMemcpyHostToDevice ); 

	//5. lunch kernel
	dim3 dimBlock(BlockDimX,BlockDimY);// 
	dim3 dimGrid(GridDimX,GridDimY);//  
	//############# convergence loop #############################
	threshold = -1;
	black_cell_count = 0;
	t = clock();
	while(black_cell_count<(75*(img.num_pixel)/100))
	{
		black_cell_count = 0;
		threshold+=1;
		// let the GPU calculate the sobel image
		sobel<<<dimGrid, dimBlock>>>(bmp_data_device,new_bmp_img_device,threshold,ht,wd);
		//copy back the black_cell_count_array from the GPU
		hipMemcpy(new_bmp_img1 ,new_bmp_img_device, img.num_pixel, hipMemcpyDeviceToHost );		
		for(int i = 1; i< (ht-1);i++)
		{
			for(int j=1;j<wd-1;j++)
				if(new_bmp_img1[i*wd+j]==0)
					black_cell_count++;
		}
		
	}
	t = clock()-t;
	printf("Time taken for CUDA sobel operation:%f sec\n",((float)t)/CLOCKS_PER_SEC);
	printf("Threshold during convergence:%d\n",threshold);
	printf("**********************************************************************\n");

	//7. free GPU memory
	hipFree( bmp_data_device );
	hipFree( new_bmp_img_device);

	//8. write back the new bmp image into output file
	FILE *output_file1;
	output_file1 = fopen(cuda_img,"wb");
	img.write_bmp_file(output_file1, new_bmp_img1);
	
	return 0;
}
