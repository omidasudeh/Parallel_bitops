
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
void print_matrix(double * A, int dim)
{
	for(int i=0;i<dim;i++)
	{
		for(int j=0;j<dim;j++)
			printf("%f\t",A[i*dim+j]);
		printf("\n");
	}
}
// the kernel
__global__ 
void mult_transpose(double *A,double *C, int dim) 
{
	// Each thread with the id = tid, will calculate the tid-th row of the result matrix
	int tid = blockIdx.x*blockDim.x+threadIdx.x; // which thread you are? what is your tid?
	// go calculate your row! (tid-th row)
	for(int j = 0;j<dim;j++)
	{
		double sum = 0;
		for(int k=0;k<dim;k++)
			sum+=(A[j*dim+k]*A[k*dim+j]);//sum+=A[j][k]*A[k][j];
		C[tid*dim+j]= sum;//C[tid][j] = sum;
	}
}
//test 
int main()
{
//############################################ Serial Code ############################################################
	printf("*****************************************************************************************\n");
	int dim = 1024;
	int num_elem = dim*dim;
	int array_size = num_elem*sizeof(double);	
	double *a = (double*) malloc(array_size);
	
	// initialize the array
	//here I just initialize the input array to all one
	for(int i = 0;i<num_elem;i++)
		a[i] = 1;
	//main loop for serial version
	double *result = (double*) malloc(array_size); // the resut array
	clock_t t;
	t = clock();
	for(int i = 0;i<dim;i++)
	{
		for(int j=0;j<dim;j++)
		{	
			double sum = 0;
			for(int k=0;k<dim;k++)
				sum+=(a[i*dim+k]*a[k*dim+j]);
			result[i*dim+j] = sum;
		}
	}
	t = clock()-t;
	printf("Time taken for serial operation:%f seconds\n",((float)t)/CLOCKS_PER_SEC);
//############################################# CUDA Code #############################################################
	// 1. define thread hirearchy
	int num_blocks = 4;
	int num_th_per_blk = 1024;// we are not violating the nvidea 1024 threads per block limitation 
	int dim1 = 4096;
	int num_elem1 = dim1*dim1;
	int array_size1 = num_elem1*sizeof(double);

	//2. Host memory allocation	
	double *a1 = (double*) malloc(array_size1);  //pointer to host version of the input array	
	double *result1 = (double*) malloc(array_size1); // the result array
	// initialize the array
	//here I just initialize the input array to all one
	for(int i = 0;i<num_elem1;i++)
		a1[i] = 1;
	//3. Device Memory allocation 
	double *ad, *C;			     //pointer to device version of the array, and the output array C
	hipMalloc( (void**)&ad, array_size1);
	hipMalloc( (void**)&C, array_size1);

	//4. copy array to the device
	hipMemcpy( ad, a1, array_size1, hipMemcpyHostToDevice ); 

	//5. lunch kernel
	dim3 dimBlock(num_th_per_blk);//  1D thread hirearchy. In this case 1024 thread per block
	dim3 dimGrid(num_blocks);//  In this 4 blocks in the grid
	t = clock();
	mult_transpose<<<dimGrid, dimBlock>>>(ad,C, dim1);
	
	//6. copy back the result
	
	hipMemcpy( result1, C, array_size1, hipMemcpyDeviceToHost ); 
	t = clock()-t;
	printf("Time taken for CUDA operation:%f seconds\n",((float)t)/CLOCKS_PER_SEC);

	//7. free GPU memory
	hipFree( ad );
	hipFree( C );
		
	printf("*****************************************************************************************\n");
	return 0;
}
